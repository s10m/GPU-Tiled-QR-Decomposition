#include "hip/hip_runtime.h"
extern "C" {
#include "../include/gpucalc.h"
}

#include <stdio.h>

#define NUMSTREAMS 128
#define TID threadIdx.x
#define CO(x,y,ldm) (((y)*ldm) + (x))

/* grid scheduler defines */
#define TLOC(x,y) (((y)*M)+(x))
#define tgrid(x,y) taskGrid[TLOC(x,y)]

#define TASK_AVAIL 0
#define TASK_NONE 1
#define TASK_DONE 2

enum Type {QRS, SAPP, QRD, DAPP};
enum Status {READY, DOING, DONE, NONE, NOTASKS};

typedef struct{
	enum Type taskType;
	int l, m, k;
	enum Status taskStatus;
	int mutex;
} Task;


/* cuda_queue defines */
#define cuda_maxqueues 1

/* Timer functions. */
#ifdef TIMERS
    #define TIMER_TIC_ND if ( threadIdx.x == 0 ) tic = clock();
    #define TIMER_TOC_ND(tid) toc = clock(); if ( threadIdx.x == 0 ) atomicAdd( &cuda_timers[tid] , ( toc > tic ) ? (toc - tic) : ( toc + (0xffffffff - tic) ) );
    #define TIMER_TIC clock_t tic; if ( threadIdx.x == 0 ) tic = clock();
    #define TIMER_TOC(tid) clock_t toc = clock(); if ( threadIdx.x == 0 ) atomicAdd( &cuda_timers[tid] , ( toc > tic ) ? (toc - tic) : ( toc + (0xffffffff - tic) ) );
    #define TIMER_TIC2_ND if ( threadIdx.x == 0 ) tic2 = clock();
    #define TIMER_TOC2_ND(tid) toc2 = clock(); if ( threadIdx.x == 0 ) atomicAdd( &cuda_timers[tid] , ( toc2 > tic2 ) ? (toc2 - tic2) : ( toc2 + (0xffffffff - tic2) ) );
    #define TIMER_TIC2 clock_t tic2; if ( threadIdx.x == 0 ) tic2 = clock();
    #define TIMER_TOC2(tid) clock_t toc2 = clock(); if ( threadIdx.x == 0 ) atomicAdd( &cuda_timers[tid] , ( toc2 > tic2 ) ? (toc2 - tic2) : ( toc2 + (0xffffffff - tic2) ) );
#else
    #define TIMER_TIC_ND
    #define TIMER_TOC_ND(tid)
    #define TIMER_TIC
    #define TIMER_TOC(tid)
    #define TIMER_TIC2
    #define TIMER_TOC2(tid)
#endif

/** Struct for a task queue. */
struct queue_cuda {
    /* Indices to the first and last elements. */
    int first, last;

    /* Number of elements in this queue. */
    volatile int count;

    /* Number of elements in the recycled list. */
    volatile int rec_count;

    /* The queue data. */
    volatile int *data;

	volatile int numIn;
};

/** Timers for the cuda parts. */
enum {
    tid_mutex = 0,
    tid_queue,
    tid_gettask,
    tid_memcpy,
    tid_update,
    tid_pack,
    tid_sort,
    tid_pair,
    tid_self,
    tid_potential,
    tid_potential4,
    tid_total,
    tid_count
    };

/* Timers. */
__device__ float cuda_timers[ tid_count ];

/* The per-SM task queues. */
//__device__ struct queue_cuda cuda_queues[ cuda_maxqueues ];
/*__constant__ int cuda_nrqueues;
__constant__ int cuda_queue_size;*/

__device__ __constant__ int cuda_queue_size;
__device__ struct queue_cuda q;
__device__ int taskupdatemutex;


/**
 * @brief Lock a device mutex.
 *
 * @param m The mutex.
 *
 * Loops until the mutex can be set. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ void cuda_mutex_lock ( volatile int *m ) {
    TIMER_TIC
    while ( atomicCAS( (int *) m , 0 , 1 ) != 0 );
    TIMER_TOC( tid_mutex )
    }


/**
 * @brief Attempt to lock a device mutex.
 *
 * @param m The mutex.
 *
 * Try to grab the mutex. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_trylock ( int *m ) {
    TIMER_TIC
    int res = atomicCAS( m , 0 , 1 ) == 0;
    TIMER_TOC( tid_mutex )
    return res;
    }


/**
 * @brief Unlock a device mutex.
 *
 * @param m The mutex.
 *
 * Does not check if the mutex had been locked.
 */

__device__ void cuda_mutex_unlock ( volatile int *m ) {
    TIMER_TIC
    atomicExch( (int *) m , 0 );
    TIMER_TOC( tid_mutex )
    }
    
    
/**
 * @brief Get a task ID from the given queue.
 *
 */
 
__device__ int cuda_queue_gettask ( void ) {

    int ind, tid = -1;
    
    /* Don't even try... */
    if ( q.rec_count == q.count )
        return -1;

    /* Get the index of the next task. */
    ind = atomicAdd( &q.first , 1 );
        
    /* Wrap the index. */
    ind %= cuda_queue_size;

    /* Loop until there is a valid task at that index, getting a task if there is one */
	
    while( q.rec_count < q.count && (tid = q.data[ind]) < 0 );

	if(tid != -1)
		q.data[ind] = -1;

    /* Return the acquired task ID. */
    return tid;
    
    }


/**
 * @brief Put a task onto the given queue.
 *
 * @param tid The task ID to add to the end of the queue.
 */
 
__device__ void cuda_queue_puttask ( int tid ) {

	int ind;

	/* Get the index of the next task. */
	ind = atomicAdd( &q.last , 1 );
	
	/* wrap index */
	ind %= cuda_queue_size;
    
	/* Wait for the slot in the queue to be empty. */
	while( q.data[ind] != -1 );
	
	/* insert the new task ID */
	q.data[ind] = tid;

	atomicAdd( (int *) &q.numIn, 1);	
}
    
    
/**
 * @brief Get a task from the given task queue.
 *
 * @return A valid task ID from the queue or -1 if the queue
 * is empty.
 *
 * Picks tasks from the queue sequentially and checks if they
 * can be computed. If not, they are returned to the queue.
 *
 * This routine blocks until a valid task is picked up, or the
 * specified queue is empty.
 */
 
__device__ int runner_cuda_gettask ( void ) {

    	int tid = -1;
    
    	TIMER_TIC
    
	/* modified because all >0 tasks in queue represent valid
	   tasks in the scheduling structure */
	tid = cuda_queue_gettask();
        
    	/* Put this task into the recycling queue, if needed. */
    	if ( tid >= 0 ) {
            atomicAdd( (int *)&q.rec_count , 1 );
		atomicAdd( (int *)&q.numIn , -1 );
        }
        
    	TIMER_TOC(tid_queue);
        
    	/* Return whatever we got. */
    	return tid;
}

__device__ void init_cuda_queue( int qlen, int totalNumTasks, volatile int *newData)
{
	int j;

	if( ( TID == 0 ) && (blockIdx.x == 0) )
	{
		q.first = 0;
		q.last = 0;
		q.rec_count = 0;
		q.count = totalNumTasks;
		q.data = newData;
		q.numIn = 0;
	}

	for( j = 0; j < qlen; j ++)
	{
		q.data[j] = -1;
	}
}

/* Insert new information into the task structure at (x,y), then place the
   newly revised index into the scheduler queue. */

__device__ void makeTask(volatile Task* taskGrid, int M, int x, int y, enum Type newType, enum Status newStatus, int newK )
{
	if( atomicCAS( (int *) &tgrid(x,y).taskStatus, (int) NONE, (int) READY) == NONE ||
		atomicCAS( (int *) &tgrid(x,y).taskStatus, (int) DONE, (int) READY ) == DONE )
	{
		tgrid(x,y).taskType = newType;
		tgrid(x,y).k = newK;
	
		cuda_queue_puttask( TLOC(x,y) );
	}
}

__device__ void init_cuda_scheduler( volatile Task* taskGrid, int M, int N)
{
	int i , j, ref;

	for(j = 0; j < M; j ++)
	{
		ref = j*M;
		for(i = 0; i < N; i ++)
		{
			taskGrid[ref].l = i;
			taskGrid[ref].m = j;
			taskGrid[ref].taskStatus = NONE;
			taskGrid[ref].k = 0;
			cuda_mutex_unlock(&taskGrid[ref].mutex);
			ref ++;
		}
	}

	makeTask( taskGrid, M, 0, 0, QRS, READY, 0 );
}

__device__ enum Type getNextType(int p, int q, int k)
{
	enum Type ret;
	if(p == k)
	{
		if(q == k)
			ret = QRS;//on diagonal
		else if(q > k)
			ret = SAPP;//on diagonal row
	}
	else if(p > k)
	{
		if(q == k)
			ret = QRD;//on diagonal column
		else if(q > k)
			ret = DAPP;//in the rest
	}

	return ret;
}

__device__ int inGrid(int M, int N, int x, int y)//1 if (x,y) in grid, 0 if not
{
	int ret = 1;

	if (x >= M)
		ret = 0;
	else if (y >= N)
		ret = 0;
	else if (x < 0)
		ret = 0;
	else if (y < 0)
		ret = 0;

	return ret;
}

//checks k equal or greater and done status
__device__ int genericdone(volatile Task* taskGrid, int M, int x, int y, int k)
{
	int ret = 0;

	if(tgrid(x,y).k == k)
	{
		if(tgrid(x,y).taskStatus == DONE)
			ret = 1;
	}
	if(tgrid(x,y).k > k)
		ret = 1;

	return ret;
}

//checks if a sqr has been performed for step k
__device__ int qrsdone(volatile Task* taskGrid, int M, int k)
{
	int ret = 0;

	if(genericdone(taskGrid, M, k, k, k))
	{
		if(tgrid(k,k).taskType == QRS)
			ret = 1;
	}

	return ret;
}

//checks if a dapp has been applied to (x,y) at step k
__device__ int dappdone(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;

	if(!inGrid(M, N, x, y))
		return 1;

	if(genericdone(taskGrid, M, x, y, k))//finished operation
	{
		if(tgrid(x,y).taskType == DAPP)//is dapp task
			ret = 1;
	}

	return ret;
}

//checks if float qr has been performed on (x,y) at step k
__device__ int qrddone(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;
	
	if(genericdone(taskGrid, M, x, y, k))//check if task finished
	{
		if(tgrid(x,y).taskType == QRD)//is qrd task
			ret = 1;
	}
	
	return ret;
}

//can always do qrs if in grid
__device__ int candoQRS(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	return inGrid(M, N, x, y);
}

//if can apply at (x,y) step k, returns 1. 0 otherwise
__device__ int candoSAPP(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;

	if (!inGrid(M, N, x - 1, y))
		return 1;

	//checkqrs(k,k)k done, check vectors are ready
	if(qrsdone(taskGrid, M, k))
	{
		//checkdapp(x,y)k-1 done//check previous step completed
		if(dappdone(taskGrid, M, N, x, y, k - 1))
			ret = 1;
	}

	return ret;
}

__device__ int candoQRD(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;
	//checkgendone(x-1,k)k done check if row above is done (qrd or qrs)
	if(genericdone(taskGrid, M, x-1, y, k))
	{
		//checkdapp(x,y)k-1 done check if dapp in place has been done
		if(k == 0)//if no previous
			ret = 1;
		else if(dappdone(taskGrid, M, N, x, y, k-1))
			ret = 1;
	}

	if(!inGrid(M, N, x, y))
		ret = 0;

	return ret;
}

__device__ int candoDAPP(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;
	//checkqrd(x,k)k done
	if(qrddone(taskGrid, M, N, x, k, k))
	{
		//checkgendone(x-1,y)k done
		if(genericdone(taskGrid, M, x-1, y, k))
		{
			//checkdapp(x,y)k-1 done
			if(k == 0)
				ret = 1;
			else if(dappdone(taskGrid, M, N, x, y, k-1))
				ret = 1;
		}
	}

	return ret;
}

/* Register the finished task as completed, then go through the possible 
   successors and add them to the task structure if it is possible to add them */
__device__ void completeATask	(volatile Task* taskGrid,
				int M, int N,
				Task t)
{
	int k, j, p, q;
	enum Type tType, tTypeNext;
	
	//cuda_mutex_lock(&tgrid(0,0).mutex);

	p = t.l;
	q = t.m;
	k = tgrid(p,q).k;
	tType = getNextType(p, q, k);

	tgrid(p,q).taskStatus = DONE;
	
	switch(tType)
	{
		case QRS:
		{
			if(candoQRD(taskGrid, M, N, p+1, q, k))//check one below
				makeTask(taskGrid, M, p+1, q, QRD, READY, k);

			for(j = k + 1; j < N; j ++)//check along row
			{
				//cuda_mutex_lock(&tgrid(p,j).mutex);

				if(candoSAPP(taskGrid, M, N, p, j, k))
					makeTask(taskGrid, M, p, j, SAPP, READY, k);

				//cuda_mutex_unlock(&tgrid(p,j).mutex);
			}
			break;
		}
		case SAPP:
		{
			if(candoDAPP(taskGrid, M, N, p+1, q, k))//check one below
				makeTask(taskGrid, M, p+1, q, DAPP, READY, k);

			break;
		}
		case QRD:
		{
			if(inGrid(M, N, p+1,q))
			{
				//cuda_mutex_lock(&tgrid(p+1,q).mutex);
				if(candoQRD(taskGrid, M, N, p+1, q, k))//check one below
					makeTask(taskGrid, M, p+1, q, QRD, READY, k);
				//cuda_mutex_unlock(&tgrid(p+1,q).mutex);
			}

			for(j = k + 1; j < N; j ++)
			{
				//cuda_mutex_lock(&tgrid(p-1,j).mutex);
				if(candoDAPP(taskGrid, M, N, p, j, k))//check along row
					makeTask(taskGrid, M, p, j, DAPP, READY, k);
				//cuda_mutex_unlock(&tgrid(p-1,j).mutex);
			}
			
			break;
		}
		case DAPP:
		{
			tTypeNext = getNextType(p, q, k + 1);

			switch(tTypeNext)//check whether can activate any for next step
			{
				case QRS:
				{
					if(candoQRS(taskGrid, M, N, p, q, k + 1))
						makeTask(taskGrid, M, p, q, QRS, READY, k + 1);
					break;
				}
				case SAPP:
				{
					if(candoSAPP(taskGrid, M, N, p, q, k + 1))
						makeTask(taskGrid, M, p, q, SAPP, READY, k + 1);

					break;
				}
				case QRD:
				{
					if(candoQRD(taskGrid, M, N, p, q, k + 1))
						makeTask(taskGrid, M, p, q, QRD, READY, k + 1);

					break;
				}
				case DAPP:
				{
					if(candoDAPP(taskGrid, M, N, p, q, k + 1))
						makeTask(taskGrid, M, p, q, DAPP, READY, k + 1);
					break;
				}
			}

			if(candoDAPP(taskGrid, M, N, p + 1, q, k))//check one below in current step
				makeTask(taskGrid, M, p + 1, q, DAPP, READY, k);

			break;
		}
	}
	//cuda_mutex_unlock(&tgrid(0,0).mutex);
}

//computes the sum of the elements of the size startN sumVector, storing the result in sumVector[0] in 5 cycles
__device__ void reduceSum(	volatile float* sumVector,
				char startN)
{
	char n = startN >> 1;

	while(n > 0)
	{
		if(TID < n)
			sumVector[TID] = sumVector[TID] + sumVector[TID + n];
		n = n >> 1;//n /= 2
	}
}	

__device__ void calcHH(	float matelem,//the block containing a column to calculate the householder vector of
			volatile float hhVector[],//the array to store the resulting vector in
			int k)//the step at which this was called
{
	float localdiv;
	int sign;

	//read vectors in from below the diagonal(k)
	if(TID >= k)
		hhVector[TID] = matelem;
	if(TID < k)//zero above diagonal
		hhVector[TID] = 0;

	//square each element
	hhVector[TID] *= hhVector[TID];
	
	//ideally only do required computation here; not necessarily 16, 8, 4, 2, 1
	//reduction to calculate the sum of squares of the 32 element vector
	reduceSum(hhVector, 32);

	//calculate sign*norm and put in local variable
	localdiv = sqrt(hhVector[0]);

	//if norm not 0
	if(localdiv != 0.0)
	{
		hhVector[TID] = matelem;

		sign = hhVector[k] >= 0 ? 1 : -1;
		
		localdiv *= sign;
		//add element in block at (k,k) to norm and store in vector
		localdiv += hhVector[k];

		//divide because want v(k+1:m) = v(k+1:m)/v(k) = v(k+1:m) * (1/v(k))
		localdiv = 1.0/localdiv;
	}
	else//if norm is zero, 
		localdiv = 1.0;

	if(TID < k)
		hhVector[TID] = 0;
	if(TID == k)
		hhVector[TID] = 1;
	if(TID > k)
		hhVector[TID] = matelem * localdiv;
}

__device__ void applyHH(volatile float blockCache[],//SHARED the 32*32 matrix block to compute over
			float* matTau, int ldm,
			volatile float workingVector[],//SHARED a working space of size 32 used in the computation
			int k,//the step at which the function was called, A(k:m, k:n) is submatrix to operate on
			float hhVectorelem)//element of vector storing the householder vector, zero above diagonal
{
	float 	y,//y will be -2 divided by the sum of squares of vector
		z;//z will be v[TID] * sum(A(:,j) .* v) * y

	int 	j;//column counter, reference in block

	//thread TID starts at (TID, k)
	//blockref = (k*32) + TID;
	
	//read data for summation
	workingVector[TID] = hhVectorelem;
	//square elements of working vector
	workingVector[TID] *= workingVector[TID];

	//reduction to find sum of squares of workingVector
	reduceSum(workingVector, 32);

	//make y a register equal to -2/sum of squares of v(v'*v)
	y = 2.0 / workingVector[0];
	
	for(j = k; j < 32; j ++)//submatrix A(k:m,k:n)
	{
		//fill workingVector with the componentwise multiplication of householder vector (zero above k) and column j of block
		workingVector[TID] = blockCache[TID+(j*32)] * hhVectorelem;

		//reduction to find sum of multiplication of column of block with hhVector
		reduceSum(workingVector, 32);
		
		//set z = TIDth element of v times -2/v'v
		z = hhVectorelem * y;

		//multiply z by sum of componentwise multiplication of column with hhVector in workingVector[0]
		z *= workingVector[0];
		
		//add z to block(TID,j). zero above diagonal
		blockCache[TID + (j*32)] -= z;
	}
	
	if(TID > k)//store essential part of vector below diagonal
		blockCache[TID + k*32] = hhVectorelem;//insert essential part of vector below diagonal in column k of block
	
	if(TID == 0)
		matTau[k] = y;
}

__device__ void calcDoubleHHWY	(float topElem,
				float lowElem,
				int k,
				volatile float blockCache[])
{
	/* Calculate v_k. */
	int sign, i;

	float 	topSum = 0,
		lowSum = 0,
		alpha;

	__shared__ volatile float first;
	
	if(TID == k)
		first = topElem;
	
	topSum = first * first;
	/* Sum squares of V to compute norm, using column of BlockCache as working space. */
	blockCache[TID + (k*32)] = lowElem * lowElem;
	for(i = 0; i < 32; i ++)
		lowSum += blockCache[i + (k*32)];
	
	alpha = sqrt(topSum + lowSum);
	sign = first >= 0 ? 1 : -1;

	if(alpha != 0.0)
	{
		/* Zeroth element is = first + sign*norm */
		alpha = first + sign * alpha;
		alpha = 1.0/alpha;
	}
	else
		alpha = 1.0;

	//topElem *= alpha;
	lowElem *= alpha;
	
	blockCache[TID + (k*32)] = lowElem;
}

__device__ void calcDoubleHH	(float topElem,//element of top block
				float lowElem,
				volatile float hhVector[], //SHARED 32x1 array to insert vector
				int k)//step at which called. use column k
{
	float tmp;//elemK not used in TID > 0
	int sign;

	if(TID < k)//zero above diagonal in top
		hhVector[TID] = 0.0;

	if(TID == k)//top nonzero element
		hhVector[k] = topElem;

	if(TID > k)//zero below diagonal in top
		hhVector[TID] = 0.0;

	//all read lower block in
	hhVector[TID + 32] = lowElem;

	if(TID == k)//kth thread holds non zero element in top block
	{
		sign = topElem >= 0 ? 1 : -1;
	}

	//all threads hold elements from bottom block
	tmp = hhVector[TID + 32];

	//square top nonzero in hhVector
	if(TID == k)
		hhVector[k] *= hhVector[k];

	//square each element in bottom block
	hhVector[TID + 32] *= hhVector[TID + 32];

	//reduce to compute sum of squares in 0th element of 64 element hhVector
	reduceSum(hhVector, 64);

	if(TID == k)
	{
		//store sign * norm in kth position
		hhVector[k] = sign * sqrt(hhVector[0]);

		if(hhVector[k] != 0.0)
		{
			//add sign*norm to kth element and store
			hhVector[k] = topElem + hhVector[k];
			
			//divide because want to divide by hhVector[k]
			hhVector[k] = 1.0/hhVector[k];
		}
		else//norm zero
			hhVector[k] = 1.0;
	}

	//normalise by multiplying by kth element
	if(TID != k)
		hhVector[32 + TID] = tmp * hhVector[k];
	if(TID == k)
		hhVector[32 + k] = tmp * hhVector[k];

	if(TID == k)//top part is 1 on diagonal
		hhVector[k] = 1.0;
	else if(TID != k)//zero elsewhere
		hhVector[TID] = 0.0;
}

__device__ void applyDoubleHH	(float topRow[],
				float lowRow[],
				float* blockTau, int ldm,
				volatile float workingVector[],
				int k,
				float hhVectorElem)
{
	float	y,//-2/v'v
		zupp, zlow;//y * v[i] *sum(A(:,j) .* v) for both blocks

	int 	j;//column counter

	//copy hhVector and square square each element for summation
	if(TID == k)
		workingVector[TID] = 1.0;
	if(TID != k)
		workingVector[TID] = 0.0;

	workingVector[TID + 32] = hhVectorElem * hhVectorElem;
	
	//reduce to find sum of squares
	reduceSum(workingVector, 64);
	
	//set y = -2/sum of squares
	y = 2 / workingVector[0];

	//for each column
	for(j = k; j < 32; j ++)
	{
		//fill working vector[i] with top block(i,j) * hhVector[i]
		if(TID == k)
			workingVector[TID] = topRow[j];
		if(TID != k)
			workingVector[TID] = 0.0;

		//fill workingVector[i + 32] with bottom block(i,j) * hhVector[i+32]
		workingVector[TID + 32] = lowRow[j] * hhVectorElem;

		//sum to find sum of componentwise multiplication
		reduceSum(workingVector, 64);
		
		//set zupp = TIDth element of hhvector times -2/v'v
		if(TID == k)
			zupp = y;
		if(TID != k)
			zupp = 0.0;

		zlow = y * hhVectorElem;

		//multiply both by sum of multiplication
		zupp *= workingVector[0];
		zlow *= workingVector[0];
		
		//add to top block element
		topRow[j] -= zupp;

		//add to bottom block element
		lowRow[j] -= zlow;
	}
	
	if(TID == 0)
		blockTau[k] = y;
	
	lowRow[k] = hhVectorElem;
}

__device__ void device_doQRS	( float* matrix, float* matTau,
				int ldm,
				volatile float workingVector[],
				volatile float blockCache[])
{
	int k, j, refCache, refMat;

	refMat = TID;
	refCache = TID;

	for(j = 0; j < 32; j ++)//load block into shared memory
	{
		blockCache[refCache] = matrix[refMat];

		refMat += ldm;
		refCache += 32;
	}

	for(k = 0; k < 32; k ++)
	{
		//calculate the kth hh vector from the kth column of the TIDth row of the matrix
		calcHH	(blockCache[TID + (k*32)],//(tid, k)
			workingVector,
			k);

		//calculate the application of the hhvector along row TID
		applyHH	(blockCache,
			matTau,
			ldm,
			workingVector,
			k,
			workingVector[TID]);
	}

	//copy row back
	refMat = TID;
	refCache = TID;

	for(j = 0; j < 32; j ++)
	{
		matrix[refMat] = blockCache[refCache];

		refMat += ldm;
		refCache += 32;
	}
	__threadfence();
}

__global__ void doQRS( float* matrix, float* tau, int ldm)
{
	__shared__ volatile float workingVector[32];
	__shared__ volatile float blockCache[32*32];
	
	device_doQRS	(matrix, tau,
			ldm,
			workingVector,
			blockCache);
}

__device__ void applyOneHHVectD	(float* topElem,
				float* lowElem,
				int k,
				volatile float tau[],
				volatile float blockCache[])
{
	float alpha;
	__shared__ volatile float workV[32];

	/* Compute alpha = sum */
	if(TID == k)
		workV[TID] = *topElem;
	if(TID != k)
		workV[TID] = 0.0;

	workV[TID] += *lowElem * blockCache[TID + (k*32)];

	reduceSum(workV, 32);
	
	alpha = workV[0];

	/* Multiply by tau */
	alpha *= tau[k];

	/* Compute alpha *= a_tid,j*v_tid,k */
	if(TID == k)
		*topElem -= alpha;
			
	/* For lower element. */
	alpha *= blockCache[TID + (k*32)];
	*lowElem -= alpha;
}

__device__ void device_doQRDW	(float* blockA, float* blockB, float* blockTau,
				int ldm,
				volatile float tauVect[],
				volatile float blockCache[])//32*32 temp space
{
	/* Idea: for each column j of (AB)^T,
		apply HH vectors 0...j-1,
		compute new HH vector for column j,
		store essential part in blockCache 

	Uses one block cache, one vector storage. */
	int j, k, i;

	float 	topElem,
		lowElem,
		tau;

	for(j = 0; j < 32; j ++)
	{
		/* Apply previous HH vectors to column j. */
		if(TID <= j)
			topElem = blockA[TID + (j*ldm)];
		if(TID > j)
			topElem = 0;

		lowElem = blockB[TID + (j*ldm)];

		for(k = 0; k < j; k ++)
		{
			/* Compute b_tid,j = b_tid,j - tau*vv'*b_tid,j */
			applyOneHHVectD	(&topElem, &lowElem,
					k,
					tauVect,
					blockCache);
		}

		calcDoubleHHWY	(topElem,
				lowElem,
				j,
				blockCache);

		/* Compute new tau = 2/v'v */
		tau = 1.0;
		for(i = 0; i < 32; i ++)
			tau += blockCache[i + (j*32)] * blockCache[i + (j*32)];
		tau = 2.0/tau;
		
		if(TID == j)
			tauVect[j] = tau;

		/* Apply new vector to column. */
		applyOneHHVectD	(&topElem, &lowElem,
				j,
				tauVect,
				blockCache);

		/* Write back */
		if(TID <= j)
			blockA[TID + (j*ldm)] = topElem;
	}

	/* Write back lower block, containing householder Vectors. */
	for(j = 0; j < 32; j ++)
		blockB[TID + (j*ldm)] = blockCache[TID + (j*32)];
	
	blockTau[TID] = tauVect[TID];
}

__device__ void device_doQRD	(float* blockA,  float* blockB, float* blockTau,
				int ldm,
				volatile float workingVector[],
				float topRow[],
				float lowRow[])
{
	int k, j, ref;
	
	ref = TID;
	for(j = 0; j < 32; j ++)//for each column
	{
		//read top block
		topRow[j] = blockA[ref];

		//read lower block into lower 32x32 square
		lowRow[j] = blockB[ref];
		ref += ldm;
	}

	for(k = 0; k < 32; k ++)
	{
		//calculate and store the vector
		calcDoubleHH	(topRow[k],
				lowRow[k],
				workingVector,
				k);

		//apply vector to both tidth rows of the matrix
		applyDoubleHH	(topRow,
				lowRow,
				blockTau,
				ldm,
				workingVector,
				k,
				workingVector[TID + 32]);
	}

	ref = TID;
	for(j = 0; j < 32; j ++)
	{
		//write back to correct blocks
		blockA[ref] = topRow[j];
		blockB[ref] = lowRow[j];
		ref += ldm;
	}
	__threadfence();
}

__global__ void doQRD( float* blockA,  float* blockB, float* blockTau, int ldm)
{
	__shared__ volatile float workingVector[64];
	
	float rowA[32], rowB[32];

	device_doQRD	(blockA, blockB, blockTau, ldm,
			workingVector,
			rowA,
			rowB);
}

__device__ void device_doSAPP	(float* blockV,
				float* blockA,
				float* blockTau,
				int ldm,
				volatile float workingVector[],
				volatile float blockCache[])
{
	int 	j, k;

	__shared__ volatile float tau[32];
	
	float 	alpha,
		belem;
	
	/* Load tau Vector */
	tau[TID] = blockTau[TID];
	
	/* Load Vectors */
	//for(j = 0; j < 32; j ++) if(TID > j) blockCache[TID + (j*32)] = blockV[TID + (j*ldm)];

	for(j = 0; j < 32; j ++)
	{
		if(TID < j)
			blockCache[TID + (j*32)] = 0.0;
		if(TID == j)
			blockCache[TID + (j*32)] = 1.0;
	}
	
	/* Compute b_j -= tau*v*v'b_j, for all vectors in blockCached V */
	for(j = 0; j < 32; j ++)
	{
		belem = blockA[TID + (j*ldm)];
		/* For each vector in block of vectors. */
		for(k = 0; k < 32; k ++)
		{
			/* Compute alpha = v'*b_j */
			workingVector[TID] = blockCache[TID + (k*32)] * belem;
			reduceSum(workingVector, 32);
			alpha = workingVector[0];

			/* Compute alpha = tau * v_tid * alpha */
			alpha *= tau[k];
			alpha *= blockCache[TID + (k*32)];
			
			/* Compute belem -= alpha */
			belem -= alpha;
		}
		blockA[TID + (j*ldm)] = belem;
	}
}

__global__ void doSAPP	(float* blockV,
			float* blockA,
			float* blockTau,
			int ldm)
{
	__shared__ volatile float workingVector[32];
	__shared__ volatile float blockCache[32*32];
	
	device_doSAPP	(blockV, blockA, blockTau,
			ldm,
			workingVector,
			blockCache);
}

__device__ void device_doDAPP	(float* blockV,
				float* blockA,
				float* blockB,
				float* blockTau,
				int ldm,
				volatile float workingVector[],
				volatile float blockCache[])
{
	__shared__ volatile float currentTau;

	float 	aelem, belem,
		tauelem,
		beta;

	int j, k, refMat, refCache;
	
	refMat = TID;
	refCache = TID;
	tauelem = blockTau[TID];

	/* Load the essential HH vector block into shared cache. */
	for(j = 0; j < 32; j ++)
	{
		blockCache[refCache] = blockV[refMat];

		refMat += ldm;
		refCache += 32;
	}

	/* For each column of the result. */
	for(j = 0; j < 32; j ++)
	{
		/* Load the elements of the vector to process. */
		aelem = blockA[TID + (j*ldm)];
		belem = blockB[TID + (j*ldm)];
		
		/* For each vector in blockV. */
		for(k = 0; k < 32; k ++)
		{
			/* Set the tau correctly. */
			if(TID == k)
				currentTau = tauelem;

			/* Compute v'*b_j */
			if(TID == k)//v_k == 1 here
				workingVector[TID] = aelem;
			if(TID != k)
				workingVector[TID] = 0;

			/* Then for lower block. */
			workingVector[TID + 32] = belem * blockCache[TID + (k*32)];
			reduceSum(workingVector, 64);

			/* Set beta = tau * v_k'b_j */
			beta = currentTau * workingVector[0];

			/* Compute b_j = b_j - beta * v_k */
			if(TID == k)//v_k is 1
				aelem -= beta;
			belem -= beta * blockCache[TID + (k*32)];
		}
		
		/* put the elements back. */
		blockA[TID + (j*ldm)] = aelem;
		blockB[TID + (j*ldm)] = belem;
	}

	/*float 	vElem,
		scal,
		mult;

	int l, j, ref;

	
	ref = TID;
	for(j = 0; j < 32; j ++)
	{
		aRow[j] = blockA[ref];
		bRow[j] = blockB[ref];
		ref += ldm;
	}

	ref = TID;
	for(l = 0; l < 32; l ++)
	{
		//load vector, ready for reduction
		vElem = blockV[ref];
		workingVector[TID] = vElem * vElem;

		//compute scal <-- -2/sum(v[tid]^2)
		//perform sum on lower half (top is I)
		reduceSum(workingVector, 32);

		scal = (-2) / (workingVector[0] + 1);

		for(j = 0; j < 32; j ++)
		{
			//sumMult <-- a_tid,j*v_tid,l
			workingVector[TID] = bRow[j] * vElem;

			if(TID == l)
				workingVector[TID] += aRow[j];

			//find for lower elements
			reduceSum(workingVector, 32);

			mult = workingVector[0] * scal;

			//a_tid,j <-- a_tid,j + scal * sumMult * v_tid,l

			if(TID == l)
				aRow[j] += mult;

			bRow[j] += mult * vElem;
		}
		ref += ldm;
	}

	ref = TID;
	for(j = 0; j < 32; j ++)
	{
		blockA[ref] = aRow[j];
		blockB[ref] = bRow[j];
		ref += ldm;
	}*/
}

__global__ void doDAPP	(float* blockV,
			float* blockA,
			float* blockB,
			float* blockTau,
			int ldm)
{
	__shared__ volatile float workingVector[32];
	__shared__ volatile float blockCache[32*32];

	/*device_doDAPP	(blockV,
			blockA,
			blockB,
			blockTau,
			ldm,
			workingVector,
			blockCache);*/
}

/* Fetch and return a task's information from the task structure, at location
   given by ref. */
inline __device__ void retrieveTask( Task* ret, volatile Task* t )
{
	//__threadfence();
	int l;

	//read and return value
	//printf("tasktzp: %p: %d\n", &ret->l, ret->l);
	l = t->l;
	
	ret->l = l;//t->l;
	ret->m = t->m;
	ret->k = t->k;
	ret->taskStatus = DOING;
	ret->taskType = t->taskType;
}

__device__ int executeTask	(Task t,
				float* mat, float* matTau,
				int ldm, int n,
				volatile float workingVector[],
				volatile float blockCache[])
{
	float *blockV, *blockA, *blockB, *blockTau;
	float blah;
	int j = 10;

	//switch based on the type of task we've got
	switch(t.taskType)
	{
		case QRS:
		{
			blockV = mat + CO(t.k*32,t.k*32,ldm);
			blockTau = matTau + CO(t.k*32,t.k*32,ldm);
			device_doQRS	( blockV, blockTau, ldm,
					workingVector, blockCache );
//			if(TID == 0)printf("%d: QRS at %d,%d\n", blockIdx.x, t.k, t.k);
			break;
		}
		case SAPP:
		{
			blockV = mat + CO(t.k*32,t.k*32,ldm);
			blockA = mat + CO(t.k*32,t.m*32,ldm);
			blockTau = matTau + CO(t.k*32,t.k*32,ldm);
			device_doSAPP	( blockV, blockA, blockTau, ldm, workingVector, blockCache );
//			if(TID == 0)printf("%d: SAPP from %d,%d to %d,%d\n", blockIdx.x, t.k, t.k, t.k, t.m);
			break;
		}
		case QRD:
		{
			blockA = mat + CO(t.k*32,t.k*32,ldm);
			blockB = mat + CO(t.l*32,t.k*32,ldm);
			blockTau = matTau + CO(t.l*32,t.k*32,ldm);
			device_doQRDW	( blockA, blockB, blockTau, ldm, workingVector, blockCache );
//			if(TID == 0)printf("%d: QRD on %d,%d; %d,%d\n", blockIdx.x, t.k, t.k, t.l, t.k);
			break;
		}
		case DAPP:
		{
			blockV = mat + CO(t.l*32,t.k*32,ldm);
			blockA = mat + CO(t.k*32,t.m*32,ldm);
			blockB = mat + CO(t.l*32,t.m*32,ldm);
			blockTau = matTau + CO(t.l*32,t.k*32,ldm);
			device_doDAPP	(blockV, blockA, blockB, blockTau, ldm,	workingVector, blockCache);
//			if(TID == 0)printf("%d: DAPP from %d,%d to %d,%d; %d,%d\n", blockIdx.x, t.l, t.k, t.k, t.m, t.l, t.m);
			break;
		}
	}
	for(j = 0; j < 32; j ++)
		blockCache[TID + (j*32)] = 0.0;

	return 1;
}

__global__ void taskKernel	(float* matrix,
				float* matTau,
				int m, int n,
				int totTasks,
				volatile Task* taskGrid,
				int M, int N )
{
	__shared__ volatile float workVector[64];
	__shared__ volatile float blockCache[32*32];
	
	int taskid;
	Task tasktz;
	__shared__ Task task;
	__shared__ int s_tid;

	/* repeat while there are still tasks undone */
	while(q.rec_count < totTasks )
	{
		/* retrieve task from the cuda queue */
		if(TID == 0)
		{
			taskid = runner_cuda_gettask();
			s_tid = taskid;
		}

		/* have finished if taskid is less than 0. Might also have invalid task */
		if(s_tid < 0)
		{
			//if(q.rec_count < totTasks)asm("trap;");
			continue;
		}

		/* get the specifics of this task from the main task structure */
		if( TID == 0 )
		{
			task.l = taskGrid[taskid].l;
			task.m = taskGrid[taskid].m;
			task.k = taskGrid[taskid].k;
			task.taskType = taskGrid[taskid].taskType;
		}

		/* perform the activity specified by the task t */
		executeTask	(task, matrix, matTau, m, n,
				workVector,
				blockCache);

		/* register task as finished in the task structure 
		At the same time, insert each newly activated task into the cuda queue */
		if( TID == 0 )
		{
			completeATask( taskGrid, M, N, task );
		}
	}
}

__global__ void cuda_initScheduling	(volatile Task* taskGrid, int p, int q,
					volatile int *newData,
					int qlen, int totTasks)
{
	init_cuda_queue( qlen, totTasks, newData );

	init_cuda_scheduler( taskGrid, p, q );
}

int calcTotalTasks(int m, int n)
{
	int ret;

	//calculate (3n^2m - n^3 + 3mn + n)/6
	ret = n;
	ret += 3*m*n;
	ret -= n*n*n;
	ret += 3*n*n*m;
	ret /= 6;

	//printf("%d,%d %d tasks\n",m, n, ret);
	return ret;
}

extern "C"
void cudaQRTask(float* mat, int m, int n)
{
	int totalTasks, p = m/32, q = n/32, queuelen = p * q + 1;
	volatile int *dev_data;
	//initialise task structure on GPU
	volatile Task* dev_taskGrid;

	enum hipError_t cuerr;
	
	hipEvent_t start, stop;

	float *dev_m, *dev_tau, time;
	
	totalTasks = calcTotalTasks( p, q );

	hipEventCreate(&start);
	hipEventCreate(&stop);

	cuerr = hipMalloc( (void**) &dev_taskGrid, p * q * sizeof(Task) );
	if(cuerr != hipSuccess)
		printf("error allocating task grid\n");
	cuerr = hipMalloc( (void**) &dev_data, queuelen * sizeof(int) );
	if(cuerr != hipSuccess)
		printf("error allocating dev queue data\n");
	cuerr = hipMalloc( (void**) &dev_m, m*n*sizeof(float) );
	if(cuerr != hipSuccess)
		printf("error allocating dev mat\n");
	cuerr = hipMalloc( (void**) &dev_tau, m*n*sizeof(float) );
	if(cuerr != hipSuccess)
		printf("error allocating tau mat\n");

	cuerr = hipMemcpy(dev_m, mat, m*n*sizeof(float), hipMemcpyHostToDevice);
	if(cuerr != hipSuccess)
		printf("error cpying dev mat\n");

	cuerr = hipMemcpyToSymbol(HIP_SYMBOL( cuda_queue_size), &queuelen, sizeof(int), 0, hipMemcpyHostToDevice );
	if(cuerr != hipSuccess)
		printf("error cpying size\n");

	hipEventRecord(start,0);
	/* initialise all structures for scheduling operations on the GPU */
	cuda_initScheduling<<<1,1>>>( 	dev_taskGrid, p, q,
					dev_data, queuelen, totalTasks );
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&time, start, stop);
	printf("Set up time taken: %f\n", time);
	
	hipEventRecord(start,0);

	taskKernel<<<p*q > 128 ? 128 : p*q,32>>>( 	dev_m, dev_tau,
							m, n, totalTasks, dev_taskGrid, p, q );

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time taken: %f\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cuerr = hipMemcpy(mat, dev_m, m*n*sizeof(float), hipMemcpyDeviceToHost);
	if(cuerr != hipSuccess)
		printf("copy back failed %d\n", cuerr);

	cuerr = hipFree(dev_m);
	if( cuerr != hipSuccess)
		printf("error freeing m %d\n", cuerr);
	cuerr = hipFree(dev_tau);
	if( cuerr != hipSuccess)
		printf("error freeing tau %d\n", cuerr);
	hipFree((Task *) dev_taskGrid);
	if( cuerr != hipSuccess)
		printf("error freeing task grid %d\n", cuerr);
	hipFree((int *) dev_data);
	if( cuerr != hipSuccess)
		printf("error freeing matrix %d\n\n", cuerr);

	if(cuerr != hipSuccess)
		hipDeviceReset();//printf("error freeing m %d\n", cuerr);
}

/*extern "C"
void cudaQRFull(float* mat, int m, int n)
{
	int i, j, k, p, q, s;
	int blockdm;

	float* dev_m, *dev_tau, *dev_K, *dev_V, *dev_A, *dev_B, *dev_T;

	hipStream_t streams[NUMSTREAMS];
	
	for(k = 0; k < NUMSTREAMS; k ++)
		hipStreamCreate(&streams[k]);

	p = m/32;
	q = n/32;

	blockdm = 32*m;//block to block dim along row

	hipMalloc((void**) &dev_m, m*n*sizeof(float));
	hipMalloc((void**) &dev_tau, m*n*sizeof(float));

	hipMemcpy(dev_m, mat, m*n*sizeof(float), hipMemcpyHostToDevice);

	dev_K = dev_m;
	dev_T = dev_tau;

	for(k = 0; k < q; k ++)
	{
		doQRS<<<1, 32, 0, streams[0]>>>(dev_K, dev_T, m);
		hipDeviceSynchronize();

		s = 1;

		dev_A = dev_K + blockdm;//one along
		for(j = k+1; j < q; j ++)
		{
			doSAPP<<<1, 32, 0, streams[s]>>>(dev_K, dev_A, m);
			
			dev_A += blockdm;//advance along row

			s ++;
			s = s % (NUMSTREAMS - 1);
		}

		dev_V = dev_K + 32;//one down from K

		for(i = k+1; i < p; i ++)
		{
			doQRD<<<1, 32, 0, streams[0]>>>(dev_K, dev_V, m);
			hipDeviceSynchronize();

			s = 0;

			dev_A = dev_K + blockdm;//one along from K			
			dev_B = dev_V + blockdm;//one along from V
			dev_T 

			for(j = k+1; j < q; j ++)
			{
				doDAPP<<<1, 32, 0, streams[s]>>>(dev_V, dev_A, dev_B, m);
				dev_A += blockdm;
				dev_B += blockdm;
				
				s ++;
				s = s % NUMSTREAMS;
			}
			dev_V += 32;//one down from previous
		}
		dev_K += blockdm + 32;//one along, one down
		hipDeviceSynchronize();
	}

	hipMemcpy(mat, dev_m, m*n*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_m);
	for(k = 0; k < NUMSTREAMS; k ++)
		hipStreamDestroy(streams[k]);
}*/
