#include "hip/hip_runtime.h"
extern "C" {
#include "../include/gpucalc.h"
}

#include <stdio.h>

#define NUMSTREAMS 128
#define TID threadIdx.x
#define CO(x,y,ldm) (((y)*ldm) + (x))

/* grid scheduler defines */
#define TLOC(x,y) (((y)*M)+(x))
#define tgrid(x,y) taskGrid[TLOC(x,y)]

#define TASK_AVAIL 0
#define TASK_NONE 1
#define TASK_DONE 2

enum Type {QRS, SAPP, QRD, DAPP};
enum Status {READY, DOING, DONE, NONE, NOTASKS};

typedef struct{
	enum Type taskType;
	int l, m, k;
	enum Status taskStatus;
	int mutex;
} Task;


/* cuda_queue defines */
#define cuda_maxqueues 1

/* Timer functions. */
#ifdef TIMERS
    #define TIMER_TIC_ND if ( threadIdx.x == 0 ) tic = clock();
    #define TIMER_TOC_ND(tid) toc = clock(); if ( threadIdx.x == 0 ) atomicAdd( &cuda_timers[tid] , ( toc > tic ) ? (toc - tic) : ( toc + (0xffffffff - tic) ) );
    #define TIMER_TIC clock_t tic; if ( threadIdx.x == 0 ) tic = clock();
    #define TIMER_TOC(tid) clock_t toc = clock(); if ( threadIdx.x == 0 ) atomicAdd( &cuda_timers[tid] , ( toc > tic ) ? (toc - tic) : ( toc + (0xffffffff - tic) ) );
    #define TIMER_TIC2_ND if ( threadIdx.x == 0 ) tic2 = clock();
    #define TIMER_TOC2_ND(tid) toc2 = clock(); if ( threadIdx.x == 0 ) atomicAdd( &cuda_timers[tid] , ( toc2 > tic2 ) ? (toc2 - tic2) : ( toc2 + (0xffffffff - tic2) ) );
    #define TIMER_TIC2 clock_t tic2; if ( threadIdx.x == 0 ) tic2 = clock();
    #define TIMER_TOC2(tid) clock_t toc2 = clock(); if ( threadIdx.x == 0 ) atomicAdd( &cuda_timers[tid] , ( toc2 > tic2 ) ? (toc2 - tic2) : ( toc2 + (0xffffffff - tic2) ) );
#else
    #define TIMER_TIC_ND
    #define TIMER_TOC_ND(tid)
    #define TIMER_TIC
    #define TIMER_TOC(tid)
    #define TIMER_TIC2
    #define TIMER_TOC2(tid)
#endif

/** Struct for a task queue. */
struct queue_cuda {
    /* Indices to the first and last elements. */
    int first, last;

    /* Number of elements in this queue. */
    volatile int count;

    /* Number of elements in the recycled list. */
    volatile int rec_count;

    /* The queue data. */
    volatile int *data;

	volatile int numIn;
};

/** Timers for the cuda parts. */
enum {
    tid_mutex = 0,
    tid_queue,
    tid_gettask,
    tid_memcpy,
    tid_update,
    tid_pack,
    tid_sort,
    tid_pair,
    tid_self,
    tid_potential,
    tid_potential4,
    tid_total,
    tid_count
    };

/* Timers. */
__device__ float cuda_timers[ tid_count ];

/* The per-SM task queues. */
//__device__ struct queue_cuda cuda_queues[ cuda_maxqueues ];
/*__constant__ int cuda_nrqueues;
__constant__ int cuda_queue_size;*/

__device__ __constant__ int cuda_queue_size;
__device__ struct queue_cuda q;
__device__ int taskupdatemutex;


/**
 * @brief Lock a device mutex.
 *
 * @param m The mutex.
 *
 * Loops until the mutex can be set. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ void cuda_mutex_lock ( volatile int *m ) {
    TIMER_TIC
    while ( atomicCAS( (int *) m , 0 , 1 ) != 0 );
    TIMER_TOC( tid_mutex )
    }


/**
 * @brief Attempt to lock a device mutex.
 *
 * @param m The mutex.
 *
 * Try to grab the mutex. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_trylock ( int *m ) {
    TIMER_TIC
    int res = atomicCAS( m , 0 , 1 ) == 0;
    TIMER_TOC( tid_mutex )
    return res;
    }


/**
 * @brief Unlock a device mutex.
 *
 * @param m The mutex.
 *
 * Does not check if the mutex had been locked.
 */

__device__ void cuda_mutex_unlock ( volatile int *m ) {
    TIMER_TIC
    atomicExch( (int *) m , 0 );
    TIMER_TOC( tid_mutex )
    }
    
    
/**
 * @brief Get a task ID from the given queue.
 *
 */
 
__device__ int cuda_queue_gettask ( void ) {

    int ind, tid = -1;
    
    /* Don't even try... */
    if ( q.rec_count == q.count )
        return -1;

    /* Get the index of the next task. */
    ind = atomicAdd( &q.first , 1 );
        
    /* Wrap the index. */
    ind %= cuda_queue_size;

    /* Loop until there is a valid task at that index, getting a task if there is one */
	
    while( q.rec_count < q.count && (tid = q.data[ind]) < 0 );

	if(tid != -1)
		q.data[ind] = -1;

    /* Return the acquired task ID. */
    return tid;
    
    }


/**
 * @brief Put a task onto the given queue.
 *
 * @param tid The task ID to add to the end of the queue.
 */
 
__device__ void cuda_queue_puttask ( int tid ) {

	int ind;

	/* Get the index of the next task. */
	ind = atomicAdd( &q.last , 1 );
	
	/* wrap index */
	ind %= cuda_queue_size;
    
	/* Wait for the slot in the queue to be empty. */
	while( q.data[ind] != -1 );
	
	/* insert the new task ID */
	q.data[ind] = tid;

	atomicAdd( (int *) &q.numIn, 1);	
}
    
    
/**
 * @brief Get a task from the given task queue.
 *
 * @return A valid task ID from the queue or -1 if the queue
 * is empty.
 *
 * Picks tasks from the queue sequentially and checks if they
 * can be computed. If not, they are returned to the queue.
 *
 * This routine blocks until a valid task is picked up, or the
 * specified queue is empty.
 */
 
__device__ int runner_cuda_gettask ( void ) {

    	int tid = -1;
    
    	TIMER_TIC
    
	/* modified because all >0 tasks in queue represent valid
	   tasks in the scheduling structure */
	tid = cuda_queue_gettask();
        
    	/* Put this task into the recycling queue, if needed. */
    	if ( tid >= 0 ) {
            atomicAdd( (int *)&q.rec_count , 1 );
		atomicAdd( (int *)&q.numIn , -1 );
        }
        
    	TIMER_TOC(tid_queue);
        
    	/* Return whatever we got. */
    	return tid;
}

__device__ void init_cuda_queue( int qlen, int totalNumTasks, volatile int *newData)
{
	int j;

	if( ( TID == 0 ) && (blockIdx.x == 0) )
	{
		q.first = 0;
		q.last = 0;
		q.rec_count = 0;
		q.count = totalNumTasks;
		q.data = newData;
		q.numIn = 0;
	}

	for( j = 0; j < qlen; j ++)
	{
		q.data[j] = -1;
	}
}

/* Insert new information into the task structure at (x,y), then place the
   newly revised index into the scheduler queue. */

__device__ void makeTask(volatile Task* taskGrid, int M, int x, int y, enum Type newType, enum Status newStatus, int newK )
{
	if( atomicCAS( (int *) &tgrid(x,y).taskStatus, (int) NONE, (int) READY) == NONE ||
		atomicCAS( (int *) &tgrid(x,y).taskStatus, (int) DONE, (int) READY ) == DONE )
	{
		tgrid(x,y).taskType = newType;
		tgrid(x,y).k = newK;
	
		cuda_queue_puttask( TLOC(x,y) );
	}
}

__device__ void init_cuda_scheduler( volatile Task* taskGrid, int M, int N)
{
	int i , j, ref;

	for(j = 0; j < M; j ++)
	{
		ref = j*M;
		for(i = 0; i < N; i ++)
		{
			taskGrid[ref].l = i;
			taskGrid[ref].m = j;
			taskGrid[ref].taskStatus = NONE;
			taskGrid[ref].k = 0;
			cuda_mutex_unlock(&taskGrid[ref].mutex);
			ref ++;
		}
	}

	makeTask( taskGrid, M, 0, 0, QRS, READY, 0 );
}

__device__ enum Type getNextType(int p, int q, int k)
{
	enum Type ret;
	if(p == k)
	{
		if(q == k)
			ret = QRS;//on diagonal
		else if(q > k)
			ret = SAPP;//on diagonal row
	}
	else if(p > k)
	{
		if(q == k)
			ret = QRD;//on diagonal column
		else if(q > k)
			ret = DAPP;//in the rest
	}

	return ret;
}

__device__ int inGrid(int M, int N, int x, int y)//1 if (x,y) in grid, 0 if not
{
	int ret = 1;

	if (x >= M)
		ret = 0;
	else if (y >= N)
		ret = 0;
	else if (x < 0)
		ret = 0;
	else if (y < 0)
		ret = 0;

	return ret;
}

//checks k equal or greater and done status
__device__ int genericdone(volatile Task* taskGrid, int M, int x, int y, int k)
{
	int ret = 0;

	if(tgrid(x,y).k == k)
	{
		if(tgrid(x,y).taskStatus == DONE)
			ret = 1;
	}
	if(tgrid(x,y).k > k)
		ret = 1;

	return ret;
}

//checks if a sqr has been performed for step k
__device__ int qrsdone(volatile Task* taskGrid, int M, int k)
{
	int ret = 0;

	if(genericdone(taskGrid, M, k, k, k))
	{
		if(tgrid(k,k).taskType == QRS)
			ret = 1;
	}

	return ret;
}

//checks if a dapp has been applied to (x,y) at step k
__device__ int dappdone(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;

	if(!inGrid(M, N, x, y))
		return 1;

	if(genericdone(taskGrid, M, x, y, k))//finished operation
	{
		if(tgrid(x,y).taskType == DAPP)//is dapp task
			ret = 1;
	}

	return ret;
}

//checks if float qr has been performed on (x,y) at step k
__device__ int qrddone(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;
	
	if(genericdone(taskGrid, M, x, y, k))//check if task finished
	{
		if(tgrid(x,y).taskType == QRD)//is qrd task
			ret = 1;
	}
	
	return ret;
}

//can always do qrs if in grid
__device__ int candoQRS(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	return inGrid(M, N, x, y);
}

//if can apply at (x,y) step k, returns 1. 0 otherwise
__device__ int candoSAPP(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;

	if (!inGrid(M, N, x - 1, y))
		return 1;

	//checkqrs(k,k)k done, check vectors are ready
	if(qrsdone(taskGrid, M, k))
	{
		//checkdapp(x,y)k-1 done//check previous step completed
		if(dappdone(taskGrid, M, N, x, y, k - 1))
			ret = 1;
	}

	return ret;
}

__device__ int candoQRD(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;
	//checkgendone(x-1,k)k done check if row above is done (qrd or qrs)
	if(genericdone(taskGrid, M, x-1, y, k))
	{
		//checkdapp(x,y)k-1 done check if dapp in place has been done
		if(k == 0)//if no previous
			ret = 1;
		else if(dappdone(taskGrid, M, N, x, y, k-1))
			ret = 1;
	}

	if(!inGrid(M, N, x, y))
		ret = 0;

	return ret;
}

__device__ int candoDAPP(volatile Task* taskGrid, int M, int N, int x, int y, int k)
{
	int ret = 0;
	//checkqrd(x,k)k done
	if(qrddone(taskGrid, M, N, x, k, k))
	{
		//checkgendone(x-1,y)k done
		if(genericdone(taskGrid, M, x-1, y, k))
		{
			//checkdapp(x,y)k-1 done
			if(k == 0)
				ret = 1;
			else if(dappdone(taskGrid, M, N, x, y, k-1))
				ret = 1;
		}
	}

	return ret;
}

/* Register the finished task as completed, then go through the possible 
   successors and add them to the task structure if it is possible to add them */
__device__ void completeATask	(volatile Task* taskGrid,
				int M, int N,
				Task t)
{
	int k, j, p, q;
	enum Type tType, tTypeNext;
	

	p = t.l;
	q = t.m;
	k = tgrid(p,q).k;
	tType = getNextType(p, q, k);

	tgrid(p,q).taskStatus = DONE;
	
	switch(tType)
	{
		case QRS:
		{
			if(candoQRD(taskGrid, M, N, p+1, q, k))//check one below
				makeTask(taskGrid, M, p+1, q, QRD, READY, k);

			for(j = k + 1; j < N; j ++)//check along row
			{
				cuda_mutex_lock(&tgrid(p,j).mutex);

				if(candoSAPP(taskGrid, M, N, p, j, k))
					makeTask(taskGrid, M, p, j, SAPP, READY, k);

				cuda_mutex_unlock(&tgrid(p,j).mutex);
			}
			break;
		}
		case SAPP:
		{
			if(candoDAPP(taskGrid, M, N, p+1, q, k))//check one below
				makeTask(taskGrid, M, p+1, q, DAPP, READY, k);

			break;
		}
		case QRD:
		{
			if(inGrid(M, N, p+1,q))
			{
				cuda_mutex_lock(&tgrid(p+1,q).mutex);
				if(candoQRD(taskGrid, M, N, p+1, q, k))//check one below
					makeTask(taskGrid, M, p+1, q, QRD, READY, k);
				cuda_mutex_unlock(&tgrid(p+1,q).mutex);
			}

			for(j = k + 1; j < N; j ++)
			{
				cuda_mutex_lock(&tgrid(p-1,j).mutex);
				if(candoDAPP(taskGrid, M, N, p, j, k))//check along row
					makeTask(taskGrid, M, p, j, DAPP, READY, k);
				cuda_mutex_unlock(&tgrid(p-1,j).mutex);
			}
			
			break;
		}
		case DAPP:
		{
			tTypeNext = getNextType(p, q, k + 1);

			switch(tTypeNext)//check whether can activate any for next step
			{
				case QRS:
				{
					if(candoQRS(taskGrid, M, N, p, q, k + 1))
						makeTask(taskGrid, M, p, q, QRS, READY, k + 1);
					break;
				}
				case SAPP:
				{
					if(candoSAPP(taskGrid, M, N, p, q, k + 1))
						makeTask(taskGrid, M, p, q, SAPP, READY, k + 1);

					break;
				}
				case QRD:
				{
					if(candoQRD(taskGrid, M, N, p, q, k + 1))
						makeTask(taskGrid, M, p, q, QRD, READY, k + 1);

					break;
				}
				case DAPP:
				{
					if(candoDAPP(taskGrid, M, N, p, q, k + 1))
						makeTask(taskGrid, M, p, q, DAPP, READY, k + 1);
					break;
				}
			}

			if(candoDAPP(taskGrid, M, N, p + 1, q, k))//check one below in current step
				makeTask(taskGrid, M, p + 1, q, DAPP, READY, k);

			break;
		}
	}
}

//computes the sum of the elements of the size startN sumVector, storing the result in sumVector[0] in 5 cycles
__device__ void reduceSum(	volatile float* sumVector,
				char startN)
{
	char n = startN >> 1;

	while(n > 0)
	{
		if(TID < n)
			sumVector[TID] = sumVector[TID] + sumVector[TID + n];
		n = n >> 1;//n /= 2
	}
}	

__device__ void calcHH(	float matelem,//the block containing a column to calculate the householder vector of
			volatile float hhVector[],//the array to store the resulting vector in
			int k)//the step at which this was called
{
	float localdiv;
	int sign;

	//read vectors in from below the diagonal(k)
	if(TID >= k)
		hhVector[TID] = matelem;
	if(TID < k)//zero above diagonal
		hhVector[TID] = 0;

	//square each element
	hhVector[TID] *= hhVector[TID];
	
	//ideally only do required computation here; not necessarily 16, 8, 4, 2, 1
	//reduction to calculate the sum of squares of the 32 element vector
	reduceSum(hhVector, 32);

	//calculate sign*norm and put in local variable
	localdiv = sqrt(hhVector[0]);

	//if norm not 0
	if(localdiv != 0.0)
	{
		hhVector[TID] = matelem;

		sign = hhVector[k] >= 0 ? 1 : -1;
		
		localdiv *= sign;
		//add element in block at (k,k) to norm and store in vector
		localdiv += hhVector[k];

		//divide because want v(k+1:m) = v(k+1:m)/v(k) = v(k+1:m) * (1/v(k))
		localdiv = 1.0/localdiv;
	}
	else//if norm is zero, 
		localdiv = 1.0;

	if(TID < k)
		hhVector[TID] = 0;
	if(TID == k)
		hhVector[TID] = 1;
	if(TID > k)
		hhVector[TID] = matelem * localdiv;
}

__device__ void applyHH(float blockRow[],// the 32*32 matrix block to compute over
			volatile float workingVector[],//SHARED a working space of size 32 used in the computation
			int k,//the step at which the function was called, A(k:m, k:n) is submatrix to operate on
			float hhVectorelem)//element of vector storing the householder vector, zero above diagonal
{
	float 	y,//y will be -2 divided by the sum of squares of vector
		z;//z will be v[TID] * sum(A(:,j) .* v) * y

	int 	j;//column counter, reference in block

	//thread TID starts at (TID, k)
	//blockref = (k*32) + TID;
	
	//read data for summation
	workingVector[TID] = hhVectorelem;
	//square elements of working vector
	workingVector[TID] *= workingVector[TID];

	//reduction to find sum of squares of workingVector
	reduceSum(workingVector, 32);

	//make y a register equal to -2/sum of squares of v(v'*v)
	y = (-2.0) / workingVector[0];
	
	for(j = k; j < 32; j ++)//submatrix A(k:m,k:n)
	{
		//fill workingVector with the componentwise multiplication of householder vector (zero above k) and column j of block
		//workingVector[TID] = matblock[blockref] * hhVectorelem;
		workingVector[TID] = blockRow[j] * hhVectorelem;

		//reduction to find sum of multiplication of column of block with hhVector
		reduceSum(workingVector, 32);
		
		//set z = TIDth element of v times -2/v'v
		z = hhVectorelem * y;

		//multiply z by sum of componentwise multiplication of column with hhVector in workingVector[0]
		z *= workingVector[0];
		
		//add z to block(TID,j). zero above diagonal
		blockRow[j] += z;

		//row major storage, next column is 32 elements ahead
		//blockref += 32;
	}
	
	if(TID > k)//store essential part of vector below diagonal
		blockRow[k] = hhVectorelem;//insert essential part of vector below diagonal in column k of block
}

__device__ void calcDoubleHH	(float topElem,//element of top block
				float lowElem,
				volatile float hhVector[], //SHARED 32x1 array to insert vector
				int k)//step at which called. use column k
{
	float tmp;//elemK not used in TID > 0
	int sign;

	if(TID < k)//zero above diagonal in top
		hhVector[TID] = 0.0;

	if(TID == k)//top nonzero element
		hhVector[k] = topElem;

	if(TID > k)//zero below diagonal in top
		hhVector[TID] = 0.0;

	//all read lower block in
	hhVector[TID + 32] = lowElem;

	if(TID == k)//kth thread holds non zero element in top block
	{
		sign = topElem >= 0 ? 1 : -1;
	}

	//all threads hold elements from bottom block
	tmp = hhVector[TID + 32];

	//square top nonzero in hhVector
	if(TID == k)
		hhVector[k] *= hhVector[k];

	//square each element in bottom block
	hhVector[TID + 32] *= hhVector[TID + 32];

	//reduce to compute sum of squares in 0th element of 64 element hhVector
	reduceSum(hhVector, 64);

	if(TID == k)
	{
		//store sign * norm in kth position
		hhVector[k] = sign * sqrt(hhVector[0]);

		if(hhVector[k] != 0.0)
		{
			//add sign*norm to kth element and store
			hhVector[k] = topElem + hhVector[k];
			
			//divide because want to divide by hhVector[k]
			hhVector[k] = 1.0/hhVector[k];
		}
		else//norm zero
			hhVector[k] = 1.0;
	}

	//normalise by multiplying by kth element
	if(TID != k)
		hhVector[32 + TID] = tmp * hhVector[k];
	if(TID == k)
		hhVector[32 + k] = tmp * hhVector[k];

	if(TID == k)//top part is 1 on diagonal
		hhVector[k] = 1.0;
	else if(TID != k)//zero elsewhere
		hhVector[TID] = 0.0;
}

__device__ void applyDoubleHH	(float topRow[],
				float lowRow[],
				volatile float workingVector[],
				int k,
				float hhVectorElem)
{
	float	y,//-2/v'v
		zupp, zlow;//y * v[i] *sum(A(:,j) .* v) for both blocks

	int 	j;//column counter

	//copy hhVector and square square each element for summation
	if(TID == k)
		workingVector[TID] = 1.0;
	if(TID != k)
		workingVector[TID] = 0.0;

	workingVector[TID + 32] = hhVectorElem * hhVectorElem;
	
	//reduce to find sum of squares
	reduceSum(workingVector, 64);
	
	//set y = -2/sum of squares
	y = (-2) / workingVector[0];

	//for each column
	for(j = k; j < 32; j ++)
	{
		//fill working vector[i] with top block(i,j) * hhVector[i]
		if(TID == k)
			workingVector[TID] = topRow[j];
		if(TID != k)
			workingVector[TID] = 0.0;

		//fill workingVector[i + 32] with bottom block(i,j) * hhVector[i+32]
		workingVector[TID + 32] = lowRow[j] * hhVectorElem;

		//sum to find sum of componentwise multiplication
		reduceSum(workingVector, 64);
		
		//set zupp = TIDth element of hhvector times -2/v'v
		if(TID == k)
			zupp = y;
		if(TID != k)
			zupp = 0.0;

		zlow = y * hhVectorElem;

		//multiply both by sum of multiplication
		zupp *= workingVector[0];
		zlow *= workingVector[0];
		
		//add to top block element
		topRow[j] += zupp;

		//add to bottom block element
		lowRow[j] += zlow;
	}
	
	lowRow[k] = hhVectorElem;
}


__device__ void applyHHPrime	(float velem,
				float applyRow[],
				int k,
				volatile float workingVector[])
{
	//apply the vector starting at row k+1 of hhVector to the (k:32) portion of applyVector
	float y, z;
	
	int j;

	//load workingVector with squares of hhVector
	workingVector[TID] = velem * velem;

	//sum to find sum of squares of hhVector
	reduceSum(workingVector, 32);

	//set y = -2/sumsquares
	y = (-2) / workingVector[0];

	//apply to columns 0:32
	for(j = 0; j < 32; j ++)
	{
		//load working vector with componentwise multiplication of hhVector with column of block A
		workingVector[TID] = velem * applyRow[j];

		//sum to find sum of pairwise mult
		reduceSum(workingVector, 32);

		//set z = sum of this times -2/sumvector
		z = y * velem;

		//multiply by element of hhVector
		z *= workingVector[0];
	
		//add to applyVector in place
		applyRow[j] += z;
	}
}

__device__ void applyDoubleHHPrime	(float velem,
					float topAppRow[],
					float lowAppRow[],
					int k,
					volatile float workingVector[])
{
	float y, z;

	int j;

	//load implied top vector
	if(TID == k)
		workingVector[k] = 1.0;
	
	if(TID != k)
		workingVector[TID] = 0.0;
	
	//square lower elements (top is 001000 so no need to multiply)
	workingVector[TID + 32] = velem * velem;

	//reduce to find sum of squares
	reduceSum(workingVector, 64);

	//set y = -2/sum of squares
	y = (-2) / workingVector[0];

	//apply to columns (0:32)
	for(j = 0; j < 32; j ++)
	{
		//load workingVector with 1 * *topAppElem (want workingVector to have component mult of hhvector and elements of application block
		if(TID == k) 
			workingVector[k] = topAppRow[j];

		if(TID != k)
			workingVector[TID] = 0.0;
	
		//do the same for the lower block but nonzero here
		workingVector[TID + 32] = velem * lowAppRow[j];

		//reduce to find sum of multiplication
		reduceSum(workingVector, 64);

		//set zTop and zLow equal to this sum times -2/v'v
		z = workingVector[0] * y;
	
		//multiply top by sum(mult) * -2/v'v. (velem here "is" 1)
		if(TID == k)
			topAppRow[j] += z;

		//multiply zLow by lower element of hhVector
		z *= velem;

		//add result to lower application element
		lowAppRow[j] += z;
	}
}
__device__ void device_doQRS	( float* matrix, int ldm,
				volatile float workingVector[],
				float bRow[])
{
	int k, j, ref;

	ref = TID;

	for(j = 0; j < 32; j ++)//load row of block into local
	{
		bRow[j] = matrix[ref];
		ref += ldm;
	}

	for(k = 0; k < 32; k ++)
	{
		//calculate the kth hh vector from the kth column of the TIDth row of the matrix
		calcHH	(bRow[k],
			workingVector,
			k);

		//calculate the application of the hhvector along row TID
		applyHH	(bRow,
			workingVector,
			k,
			workingVector[TID]);
	}

	//copy row back
	ref = TID;
	for(j = 0; j < 32; j ++)
	{
		matrix[ref] = bRow[j];
		ref += ldm;
	}
}
__global__ void doQRS( float* matrix, int ldm)
{
	__shared__ volatile float workingVector[32];
	
	float row[32];

	device_doQRS	(matrix, ldm,
			workingVector,
			row);
}

__device__ void device_doQRD	( float* blockA,  float* blockB, int ldm,
				volatile float workingVector[],
				float topRow[],
				float lowRow[])
{
	int k, j, ref;
	
	ref = TID;
	for(j = 0; j < 32; j ++)//for each column
	{
		//read top block
		topRow[j] = blockA[ref];

		//read lower block into lower 32x32 square
		lowRow[j] = blockB[ref];
		ref += ldm;
	}

	for(k = 0; k < 32; k ++)
	{
		//calculate and store the vector
		calcDoubleHH	(topRow[k],
				lowRow[k],
				workingVector,
				k);

		//apply vector to both tidth rows of the matrix
		applyDoubleHH	(topRow,
				lowRow,
				workingVector,
				k,
				workingVector[TID + 32]);
	}

	ref = TID;
	for(j = 0; j < 32; j ++)
	{
		//write back to correct blocks
		blockA[ref] = topRow[j];
		blockB[ref] = lowRow[j];
		ref += ldm;
	}
}

__global__ void doQRD( float* blockA,  float* blockB, int ldm)
{
	__shared__ volatile float workingVector[64];
	
	float rowA[32], rowB[32];

	device_doQRD	(blockA, blockB, ldm,
			workingVector,
			rowA,
			rowB);
}

__device__ void device_doSAPP	( float* blockV,
				 float* blockA,
				int ldm,
				volatile float workingVector[],
				float aRow[],
				float vRow[])
{
	float 	scal,
		mult;

	int j, l, ref;

	ref = TID;
	for(j = 0; j < 32; j ++)
	{
		aRow[j] = blockA[ref];

		if(TID < j)
			vRow[j] = 0.0;
		else if(TID == j)
			vRow[j] = 1.0;
		else if(TID > j)
			vRow[j] = blockV[ref];
		ref += ldm;
	}

	for(l = 0; l < 32; l ++)
	{
		//load vector, ready for reduction
		workingVector[TID] = vRow[l] * vRow[l];

		//compute scal <-- -2/sum(v[tid]^2)
		reduceSum(workingVector, 32);

		scal = (-2) / workingVector[0];

		for(j = 0; j < 32; j ++)
		{
			//sumMult <-- a_tid,j*v_tid,l
			workingVector[TID] = aRow[j] * vRow[l];

			//find for lower elements
			reduceSum(workingVector, 32);

			mult = workingVector[0] * scal;

			//a_tid,j <-- a_tid,j + scal * sumMult * v_tid,l
			aRow[j] += vRow[l] * mult;
		}
	}

	ref = TID;

	for(j = 0; j < 32; j ++)
	{
		blockA[ref] = aRow[j];
		ref += ldm;
	}
}

__global__ void doSAPP	( float* blockV,
			 float* blockA,
			int ldm)
{
	__shared__ volatile float workingVector[32];
	
	float aRow[32], vRow[32];

	device_doSAPP	(blockV, blockA, ldm,
			workingVector,
			aRow,
			vRow);
}

__device__ void device_doDAPP	( float* blockV,
				 float* blockA,
				 float* blockB,
				int ldm,
				volatile float workingVector[],
				float aRow[],
				float bRow[])
{
	float 	vElem,
		scal,
		mult;

	int l, j, ref;

	
	ref = TID;
	for(j = 0; j < 32; j ++)
	{
		aRow[j] = blockA[ref];
		bRow[j] = blockB[ref];
		ref += ldm;
	}

	ref = TID;
	for(l = 0; l < 32; l ++)
	{
		//load vector, ready for reduction
		vElem = blockV[ref];
		workingVector[TID] = vElem * vElem;

		//compute scal <-- -2/sum(v[tid]^2)
		//perform sum on lower half (top is I)
		reduceSum(workingVector, 32);

		scal = (-2) / (workingVector[0] + 1);

		for(j = 0; j < 32; j ++)
		{
			//sumMult <-- a_tid,j*v_tid,l
			workingVector[TID] = bRow[j] * vElem;

			if(TID == l)
				workingVector[TID] += aRow[j];

			//find for lower elements
			reduceSum(workingVector, 32);

			mult = workingVector[0] * scal;

			//a_tid,j <-- a_tid,j + scal * sumMult * v_tid,l

			if(TID == l)
				aRow[j] += mult;

			bRow[j] += mult * vElem;
		}
		ref += ldm;
	}

	ref = TID;
	for(j = 0; j < 32; j ++)
	{
		blockA[ref] = aRow[j];
		blockB[ref] = bRow[j];
		ref += ldm;
	}
}

__global__ void doDAPP	( float* blockV,
			 float* blockA,
			 float* blockB,
			int ldm)
{
	__shared__ volatile float workingVector[32];

	float 	aRow[32],
		bRow[32];

	device_doDAPP	(blockV,
			blockA,
			blockB,
			ldm,
			workingVector,
			aRow,
			bRow);
}

/* Fetch and return a task's information from the task structure, at location
   given by ref. */
inline __device__ void retrieveTask( Task* ret, volatile Task* t )
{
	//__threadfence();

	//read and return value
	t->taskStatus = DOING;
	//printf("tasktzp: %p: %d\n", &ret->l, ret->l);
	ret->l = t->l;
	ret->m = t->m;
	ret->k = t->k;
	ret->taskStatus = DOING;
	ret->taskType = t->taskType;
}

__device__ int executeTask	( Task t,
				float* mat, int ldm, int n,
				volatile float workingVector[],
				float vElems[], float topRow[], float lowRow[])
{
	 float *blockV, *blockA, *blockB;

	//switch based on the type of task we've got
	switch(t.taskType)
	{
		case QRS:
		{
			blockV = mat + CO(t.k*32,t.k*32,ldm);
			device_doQRS	( blockV, ldm,
					workingVector, topRow );
//			if(TID == 0)printf("%d: QRS at %d,%d\n", blockIdx.x, t.k, t.k);
			break;
		}
		case SAPP:
		{
			blockV = mat + CO(t.k*32,t.k*32,ldm);
			blockA = mat + CO(t.k*32,t.m*32,ldm);
			device_doSAPP	( blockV, blockA, ldm,
					workingVector, vElems, topRow );
//			if(TID == 0)printf("%d: SAPP from %d,%d to %d,%d\n", blockIdx.x, t.k, t.k, t.k, t.m);
			break;
		}
		case QRD:
		{
			blockA = mat + CO(t.k*32,t.k*32,ldm);
			blockB = mat + CO(t.l*32,t.k*32,ldm);
			device_doQRD	( blockA, blockB, ldm,
					workingVector, topRow, lowRow );
//			if(TID == 0)printf("%d: QRD on %d,%d; %d,%d\n", blockIdx.x, t.k, t.k, t.l, t.k);
			break;
		}
		case DAPP:
		{
			blockV = mat + CO(t.l*32,t.k*32,ldm);
			blockA = mat + CO(t.k*32,t.m*32,ldm);
			blockB = mat + CO(t.l*32,t.m*32,ldm);
			device_doDAPP	(blockV, blockA, blockB, ldm,
					workingVector, topRow, lowRow );
//			if(TID == 0)printf("%d: DAPP from %d,%d to %d,%d; %d,%d\n", blockIdx.x, t.l, t.k, t.k, t.m, t.l, t.m);

			break;
		}
	}

	return 1;
}

__global__ void taskKernel	(  float* matrix, int m, int n,
				int totTasks,
				volatile Task* taskGrid,
				int M, int N )
{
	__shared__ volatile float workVector[64];
	
	float 	vElems[32],
		topRow[32],
		lowRow[32];

	int taskid;
	Task tasktz;
	__shared__ Task task;
	__shared__ int s_tid;

	/* repeat while there are still tasks undone */
	while(q.rec_count < totTasks )
	{
		/* retrieve task from the cuda queue */
		if(TID == 0)
		{
			taskid = runner_cuda_gettask();
			s_tid = taskid;
			//printf("%d: got task %d\n", blockIdx.x, taskid);
		}

		/* have finished if taskid is less than 0. Might also have invalid task */
		if(s_tid < 0)
		{
			//if(TID == 0)printf("missed ");
			if(q.rec_count < totTasks)asm("trap;");
			continue;
		}

		/* get the specifics of this task from the main task structure */
		if( TID == 0 )
		{
			//printf("tasktzp: %p: %d\n", &tasktz.l, tasktz.l);
			
			retrieveTask( &tasktz, taskGrid + taskid );
			task = tasktz;
		}

		//__threadfence();
		/* perform the activity specified by the task t */
		executeTask	(task, matrix, m, n,
				workVector,
				vElems,
				topRow, lowRow);

		/* register task as finished in the task structure 
		At the same time, insert each newly activated task into the cuda queue */
		//__threadfence();
		if( TID == 0 )
		{
			completeATask( taskGrid, M, N, task );
		}
	}
	//if( (TID == 0) && (blockIdx.x == 0) )	printf("%d, %d\n", q.rec_count, q.numIn);
}

__global__ void cuda_initScheduling	(volatile Task* taskGrid, int p, int q,
					volatile int *newData,
					int qlen, int totTasks)
{
	init_cuda_queue( qlen, totTasks, newData );

	init_cuda_scheduler( taskGrid, p, q );
}

int calcTotalTasks(int m, int n)
{
	int ret;

	//calculate (3n^2m - n^3 + 3mn + n)/6
	ret = n;
	ret += 3*m*n;
	ret -= n*n*n;
	ret += 3*n*n*m;
	ret /= 6;

	//printf("%d,%d %d tasks\n",m, n, ret);
	return ret;
}

extern "C"
void cudaQRTask(float* mat, int m, int n)
{
	int totalTasks, p = m/32, q = n/32, queuelen = p * q;
	volatile int *dev_data;
	//initialise task structure on GPU
	volatile Task* dev_taskGrid;

	enum hipError_t cuerr;

	float *dev_m;
	
	totalTasks = calcTotalTasks( p, q );
	//queuelen = totalTasks + 5;

	cuerr = hipMalloc( (void**) &dev_taskGrid, p * q * sizeof(Task) );
	if(cuerr != hipSuccess)
		printf("error allocating task grid\n");		
	cuerr = hipMalloc( (void**) &dev_data, queuelen * sizeof(int) );
	if(cuerr != hipSuccess)
		printf("error allocating dev queue data\n");
	cuerr = hipMalloc( (void**) &dev_m, m*n*sizeof(float));
	if(cuerr != hipSuccess)
		printf("error allocating dev mat\n");

	cuerr = hipMemcpy(dev_m, mat, m*n*sizeof(float), hipMemcpyHostToDevice);
	if(cuerr != hipSuccess)
		printf("error cpying dev mat\n");

	cuerr = hipMemcpyToSymbol(HIP_SYMBOL( cuda_queue_size), &queuelen, sizeof(int), 0, hipMemcpyHostToDevice );
	if(cuerr != hipSuccess)
		printf("error cpying size\n");

	/* initialise all structures for scheduling operations on the GPU */
	cuda_initScheduling<<<1,1>>>( 	dev_taskGrid, p, q,
					dev_data, queuelen, totalTasks );
	taskKernel<<<128 < p*q ? 128 : p*q -1,32>>>( dev_m, m, n, totalTasks, dev_taskGrid, p, q );

	cuerr = hipMemcpy(mat, dev_m, m*n*sizeof(float), hipMemcpyDeviceToHost);
	if(cuerr != hipSuccess)
		printf("copy back failed %d\n", cuerr);

	cuerr = hipFree(dev_m);
	if( cuerr != hipSuccess)
		printf("error freeing m %d\n", cuerr);
	hipFree((Task *) dev_taskGrid);
	if( cuerr != hipSuccess)
		printf("error freeing task grid %d\n", cuerr);
	hipFree((int *) dev_data);
	if( cuerr != hipSuccess)
		printf("error freeing matrix %d\n\n", cuerr);

	if(cuerr != hipSuccess)
		hipDeviceReset();//printf("error freeing m %d\n", cuerr);
}

extern "C"
void cudaQRFull(float* mat, int m, int n)
{
	int i, j, k, p, q, s;
	int blockdm;

	float* dev_m, *dev_K, *dev_V, *dev_A, *dev_B;

	hipStream_t streams[NUMSTREAMS];
	
	for(k = 0; k < NUMSTREAMS; k ++)
		hipStreamCreate(&streams[k]);

	p = m/32;
	q = n/32;

	blockdm = 32*m;//block to block dim along row

	hipMalloc((void**) &dev_m, m*n*sizeof(float));
	hipMemcpy(dev_m, mat, m*n*sizeof(float), hipMemcpyHostToDevice);

	dev_K = dev_m;

	for(k = 0; k < q; k ++)
	{
		doQRS<<<1, 32, 0, streams[0]>>>(dev_K, m);
		hipDeviceSynchronize();

		s = 1;

		dev_A = dev_K + blockdm;//one along
		for(j = k+1; j < q; j ++)
		{
			doSAPP<<<1, 32, 0, streams[s]>>>(dev_K, dev_A, m);
			
			dev_A += blockdm;//advance along row

			s ++;
			s = s % (NUMSTREAMS - 1);
		}

		dev_V = dev_K + 32;//one down from K

		for(i = k+1; i < p; i ++)
		{
			doQRD<<<1, 32, 0, streams[0]>>>(dev_K, dev_V, m);
			hipDeviceSynchronize();

			s = 0;

			dev_A = dev_K + blockdm;//one along from K			
			dev_B = dev_V + blockdm;//one along from V

			for(j = k+1; j < q; j ++)
			{
				doDAPP<<<1, 32, 0, streams[s]>>>(dev_V, dev_A, dev_B, m);
				dev_A += blockdm;
				dev_B += blockdm;
				
				s ++;
				s = s % NUMSTREAMS;
			}
			dev_V += 32;//one down from previous
		}
		dev_K += blockdm + 32;//one along, one down
		hipDeviceSynchronize();
	}

	hipMemcpy(mat, dev_m, m*n*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_m);
	for(k = 0; k < NUMSTREAMS; k ++)
		hipStreamDestroy(streams[k]);
}
